#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
void checkCUDAError(const char *msg);
 __global__ void Kernel(int *dat);
extern "C" void  cumain(int myid, int gx, int gy,int bx, int by, int bz) {
	int *dat_local, *dat_remote;
	//int gx,gy;
	//int bx,by,bz;
	int size;
	int numthreads,j;
        int mydev;

	hipSetDevice(myid);
	hipGetDevice(&mydev);
	
	dim3 dimGrid(gx,gy);
	dim3 dimBlock(bx,by,bz);
	
	numthreads=gx*gy*bx*by*bz;
	
	size=6*sizeof(int)*numthreads;
	hipMalloc((void**) &dat_remote, size);
        checkCUDAError("hipMalloc");
	dat_local=(int*)malloc(size);
	
	Kernel<<<dimGrid,dimBlock>>>(dat_remote);
        checkCUDAError("Kernel");
	hipMemcpy(dat_local, dat_remote, size,hipMemcpyDeviceToHost);
        checkCUDAError("copy");
	
	for(int i=0;i<numthreads;i++) {
		j=i*6;
		printf("%4.4d %2.2d %6d      %3d %3d      %3d %3d %3d\n",myid,mydev,
		dat_local[j],
		dat_local[j+1],dat_local[j+2],
		dat_local[j+3],dat_local[j+4],dat_local[j+5]);
	}
}

// To run at normal speed define SLOW as blank.
// The program should return the same results independent of
// the setting for kmax and jmax.
// Set jmax to a larger value to slow it down more.
#ifndef SLOW
#define SLOW slow 
#endif
int __device__ slow(int input){
  int i;
  int jmax,kmax;
  jmax=5;
  kmax=100000;
  for (int j=1; j <=jmax ; j++) {
    i=j;
    if(j == jmax)i=input;
    for (int k=1; k< kmax; k++) {
     i= int(i*(1.00001*(sin((double)i)*sin((double)i)+cos((double)i)*cos((double)i))));
    }
  }
  return(i);
}

 __global__ void Kernel(int *dat) {
/* get my block within a grid */
    int myblock=blockIdx.x+blockIdx.y*gridDim.x;
/* how big is each block within a grid */
    int blocksize=blockDim.x*blockDim.y*blockDim.z;
/* get thread within a block */
    int subthread=threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
/* find my thread */
    int thread=myblock*blocksize+subthread;
#if __DEVICE_EMULATION__
	printf("gridDim=(%3d %3d) blockIdx=(%3d %3d)     blockDim=(%3d %3d %3d)  threadIdx=(%3d %3d %3d)  %6d\n",    
	  gridDim.x,gridDim.y,
	  blockIdx.x,blockIdx.y,
	  blockDim.x,blockDim.y,blockDim.z,
	  threadIdx.x,threadIdx.y,threadIdx.z,thread);
#endif
/* starting index into array */
	int index=thread*6;
	dat[index]=SLOW(thread);
	dat[index+1]=SLOW(blockIdx.x);
	dat[index+2]=SLOW(blockIdx.y);
	dat[index+3]=SLOW(threadIdx.x);
	dat[index+4]=SLOW(threadIdx.y);
	dat[index+5]=SLOW(threadIdx.z);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
