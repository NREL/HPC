//from https://github.com/NVIDIA/CUDALibrarySamples.git cuSOLVER/getrf

/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */
 
#ifndef pmax
int pmax=10;
#endif

#ifdef MINE
    const int m = MINE;
#else
    const int m = 45000;
#endif


#include <cstdio>
#include <cstdlib>
#include <vector>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"
double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}


int main(int argc, char *argv[]) {
    double todev1,todev2,inv1,inv2,frdev1,frdev2;
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    int device;
    int GPU=0;
    if (argc == 2) GPU = std::atoi(argv[1]);
    const int lda = m;
    const int ldb = m;

    /*       
     *       | 1 2 3  |
     *   A = | 4 5 6  |
     *       | 7 8 10 |
     *
     * without pivoting: A = L*U
     *       | 1 0 0 |      | 1  2  3 |
     *   L = | 4 1 0 |, U = | 0 -3 -6 |
     *       | 7 2 1 |      | 0  0  1 |
     *
     * with pivoting: P*A = L*U
     *       | 0 0 1 |
     *   P = | 1 0 0 |
     *       | 0 1 0 |
     *
     *       | 1       0     0 |      | 7  8       10     |
     *   L = | 0.1429  1     0 |, U = | 0  0.8571  1.5714 |
     *       | 0.5714  0.5   1 |      | 0  0       -0.5   |
     */

    // const std::vector<double> A = {1.0, 4.0, 7.0, 2.0, 5.0, 8.0, 3.0, 6.0, 10.0};
    std::vector<double> A(m*m, 0.2); 
    //const std::vector<double> B = {1.0, 2.0, 3.0};
    std::vector<double> B(m, 1); 
    std::vector<double> X(m, 0);
    std::vector<double> LU(lda * m, 0);
    std::vector<int> Ipiv(m, 0);
    int info = 0;
    int one;
    for (one=0 ; one < m*m; one=one+(m+1)) {
	    A[one]=1;
    }

    double *d_A = nullptr; /* device copy of A */
    double *d_B = nullptr; /* device copy of B */
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */

    const int pivot_on = 1;
    
    hipDeviceProp_t prop;
    hipSetDevice(GPU);
    hipGetDeviceProperties(&prop, GPU);
    hipGetDevice(&device);
    printf("On GPU %s # %d\n",prop.name,device);
    printf("Matrix size %d\n",m);
    if (pmax < m) printf("Printing %d elements\n",pmax);


    if (pivot_on) {
        printf("pivot is on : compute P*A = L*U \n");
    } else {
        printf("pivot is off: compute A = L*U (not numerically stable)\n");
    }
    if (m < pmax) {
      printf("A = (matlab base-1)\n");
      print_matrix(m, m, A.data(), lda);
    }
    else {
      printf("top corner of A = (matlab base-1)\n");
      print_matrix(pmax,pmax, A.data(), lda);
    }
    printf("=====\n");

    if (m < pmax) {
	    printf("B = (matlab base-1)\n");
	    print_matrix(m, 1, B.data(), ldb);
    }
    else {
	    printf("Beginnig of B = (matlab base-1)\n"); 
	    print_matrix(pmax, 1, B.data(), ldb);
    }
    printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */

    todev1=mysecond();
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int) * Ipiv.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(
        hipMemcpyAsync(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice, stream));
    todev2=mysecond();

    /* step 3: query working space of getrf */
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    inv1=mysecond();
    /* step 4: LU factorization */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, NULL, d_info));
    }
    inv2=mysecond();
    frdev1=mysecond();
    if (pivot_on) {
        CUDA_CHECK(hipMemcpyAsync(Ipiv.data(), d_Ipiv, sizeof(int) * Ipiv.size(),
                                   hipMemcpyDeviceToHost, stream));
    }
    CUDA_CHECK(
        hipMemcpyAsync(LU.data(), d_A, sizeof(double) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    frdev2=mysecond();
    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    if (pivot_on) {
        if (m < pmax) {
        	printf("pivoting sequence, matlab base-1\n");
        	for (int j = 0; j < m; j++) {
            	printf("Ipiv(%d) = %d\n", j + 1, Ipiv[j]);
            }

        }
        else {
			printf("beginning of pivoting sequence, matlab base-1\n");
				for (int j = 0; j < pmax; j++) {
			printf("Ipiv(%d) = %d\n", j + 1, Ipiv[j]);
			}        	
        }
    }
	if (m < pmax) {
		printf("L and U = (matlab base-1)\n");
		print_matrix(m, m, LU.data(), lda);
	
	}
	else {
		printf("top corner of L and U = (matlab base-1)\n");
		print_matrix(pmax, pmax, LU.data(), lda);
	}        	
        

    printf("=====\n");

    /*
     * step 5: solve A*X = B
     *       | 1 |       | -0.3333 |
     *   B = | 2 |,  X = |  0.6667 |
     *       | 3 |       |  0      |
     *
     */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, d_Ipiv, d_B, ldb, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, NULL, d_B, ldb, d_info));
    }

    CUDA_CHECK(
        hipMemcpyAsync(X.data(), d_B, sizeof(double) * X.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    printf("=====\n");
    if (m < pmax) {
		printf("X = (matlab base-1)\n");
		print_matrix_l(m, 1, X.data(), ldb);
	}
	else {
		printf("Beginnig of X = (matlab base-1)\n"); 
		print_matrix_l(pmax, 1, X.data(), ldb);
	}


    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_Ipiv));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());
    printf("TIMINGS (seconds)\n");
	    printf("  to device %g\n",todev2-todev1);
	    printf("     solve  %g\n",inv2-inv1);
	    printf("from device %g\n",frdev2-frdev1);
	    printf("  GPU total %g\n",frdev2-todev1);

    return EXIT_SUCCESS;
}
